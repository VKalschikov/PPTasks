#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <locale.h>
#include <stdlib.h>

#define N 2000
#define inf 1000000
#define div 200

__global__ void floydCycle(int* b, int i) {
	int k = blockIdx.x*(N/div)+threadIdx.x;
	int j = blockIdx.y*(N/div)+threadIdx.y;
	int v1 = b[j * N + k];
	int v2 = b[j * N + i] + b[i * N + k];
	if (v1 > v2) {
		b[j * N + k] = v2;
	}
}

int main()
{
	// 3 ������� A,B,C  C=A+B    NxN
	// ������ ���� ��������� 1 ������� �� C - ����� N^2


	int* a, * b;
	a = new int[N * N];
	b = new int[N * N];
	for (int i = 0; i < N; ++i) {
		for (int j = i; j < N; ++j) {
			if (i == j) {
				a[i * N + j] = 0;
			}
			else {
				if (rand() % 100 > 65) {
					a[i * N + j] = a[j * N + i] = b[i * N + j] = b[j * N + i] = rand() % 100;
				}
				else {
					a[i * N + j] = a[j * N + i] = -1;
					b[i * N + j] = b[j * N + i] = inf;
				}
			}
		}
	}

	int* dev_b;
	hipError_t cudaStatus;
	hipMalloc((void**)&dev_b, N * N * sizeof(int));

	hipError_t error;


	error = hipMemcpy(dev_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	// ����� �������� �����������

	dim3 grid(div, div);
	dim3 blocks(N/div, N/div);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// ������� ���� ��������� ������-�������
	for (int i = 0; i < N; ++i) {
		floydCycle << <grid, blocks >> > (dev_b, i);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}
	hipDeviceSynchronize();

	error = hipMemcpy(b, dev_b, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}
	/*for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%d ", a[i * N + j]);
		}
		printf("\n");
	}
	printf("=======================================\n");
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%d ", b[i * N + j]);
		}
		printf("\n");
	}*/
	printf("%f milliseconds\n", milliseconds);
	delete a;
	delete b;

	hipFree(dev_b);
	return 0;
}