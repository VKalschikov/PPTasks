#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <locale.h>
#include <stdlib.h>

#define N 2000
#define div 200

__global__ void mulmat(int* a, int *b, int *c, int i) {
	int k = blockIdx.x * (N / div) + threadIdx.x;
	int j = blockIdx.y * (N / div) + threadIdx.y;
	int a_var = a[k * N + i];
	int b_var = b[i * N + j];
	c[k * N + j] += a_var * b_var;
}

int main()
{


	int* a, * b, * c;
	a = new int[N * N];
	b = new int[N * N];
	c = new int[N * N];
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			a[i * N + j] = rand() % 10;
			b[i * N + j] = rand() % 10;
			c[i * N + j] = 0;
		}
	}

	int* dev_a, * dev_b, * dev_c;
	hipError_t cudaStatus;
	hipMalloc((void**)&dev_a, N * N * sizeof(int));
	hipMalloc((void**)&dev_b, N * N * sizeof(int));
	hipMalloc((void**)&dev_c, N * N * sizeof(int));

	hipError_t error;

	error = hipMemcpy(dev_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}
	error = hipMemcpy(dev_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}
	error = hipMemcpy(dev_c, c, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	dim3 grid(div, div);
	dim3 blocks(N / div, N / div);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	for (int i = 0; i < N; ++i) {
		mulmat << <grid, blocks >> > (dev_a, dev_b, dev_c, i);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}
	hipDeviceSynchronize();

	error = hipMemcpy(a, dev_a, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy(b, dev_b, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy(c, dev_c, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}
	/*for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%d ", a[i * N + j]);
		}
		printf("\n");
	}
	printf("=======================================\n");
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%d ", b[i * N + j]);
		}
		printf("\n");
	}*/
	printf("%f milliseconds\n", milliseconds);

	/*for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			printf("%d ", a[i*N+j]);
		}
		printf("\n");
	}
	printf("---------------------------------------------------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			printf("%d ", b[i * N + j]);
		}
		printf("\n");
	}
	printf("---------------------------------------------------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			printf("%d ", c[i * N + j]);
		}
		printf("\n");
	}*/

	delete a;
	delete b;
	delete c;

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
